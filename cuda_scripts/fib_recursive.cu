
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel for recursive Fibonacci with exactly the same pattern as your Bend code
__device__ unsigned int fib_recursive_device(unsigned int n) {
    if (n == 0) return 0;
    if (n == 1) return 1;
    // Matching the exact pattern in your Bend code: fib_recursive(n-2) + fib_recursive(n-2 + 1)
    return fib_recursive_device(n-2) + fib_recursive_device(n-2 + 1);
}

__global__ void fib_recursive_kernel(unsigned int n, unsigned int *result) {
    *result = fib_recursive_device(n);
}

int main(int argc, char *argv[]) {
    // Print the number of arguments received
    printf("Number of arguments: %d\n", argc);
    
    // Default to calculating Fibonacci of 5 if no argument is provided
    unsigned int n = 5;  // Default to a smaller number for testing
    
    if (argc > 1) {
        n = atoi(argv[1]);
        printf("Argument received: '%s', converted to: %u\n", argv[1], n);
    } else {
        printf("No argument received, using default value: %u\n", n);
    }
    
    // Allocate memory on the host and device
    unsigned int *d_result;
    unsigned int h_result;
    
    hipMalloc((void**)&d_result, sizeof(unsigned int));
    
    // Initialize device memory to a known value
    unsigned int init_value = 0;
    hipMemcpy(d_result, &init_value, sizeof(unsigned int), hipMemcpyHostToDevice);
    
    // Measure time for both methods
    clock_t start, end;
    double cpu_time_used;
    
    // Run recursive method first to match the Bend program
    start = clock();
    fib_recursive_kernel<<<1, 1>>>(n, d_result);
    hipError_t err = hipDeviceSynchronize();
    end = clock();
    
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    
    hipMemcpy(&h_result, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    
    printf("Fibonacci(%u) = %u (Recursive method)\n", n, h_result);
    printf("Time taken: %f seconds\n", cpu_time_used);
    
    // Clean up
    hipFree(d_result);
    
    return 0;
}
