
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel for recursive Fibonacci with exactly the same pattern as your Bend code
__device__ unsigned int fib_recursive_device(unsigned int n) {
    if (n == 0) return 0;
    if (n == 1) return 1;
    // Matching the exact pattern in your Bend code: fib_recursive(n-2) + fib_recursive(n-2 + 1)
    return fib_recursive_device(n-2) + fib_recursive_device(n-2 + 1);
}

__global__ void fib_recursive_kernel(unsigned int n, unsigned int *result) {
    *result = fib_recursive_device(n);
}

// CUDA kernel for iterative Fibonacci
__global__ void fib_iterative_kernel(unsigned int n, unsigned int *result) {
    if (n <= 1) {
        *result = n;
        return;
    }
    
    unsigned int a = 0, b = 1, c;
    for (unsigned int i = 2; i <= n; i++) {
        c = a + b;
        a = b;
        b = c;
    }
    
    *result = b;
}

int main(int argc, char *argv[]) {
    // Default to calculating Fibonacci of 5 if no argument is provided
    unsigned int n = 43;
    if (argc > 1) {
        n = atoi(argv[1]);
    }
    
    // Allocate memory on the host and device
    unsigned int *d_result;
    unsigned int h_result;
    
    hipMalloc((void**)&d_result, sizeof(unsigned int));
    
    // Measure time for both methods
    clock_t start, end;
    double cpu_time_used;
    
    // Run recursive method first to match the Bend program
    start = clock();
    fib_recursive_kernel<<<1, 1>>>(n, d_result);
    hipDeviceSynchronize();
    end = clock();
    
    hipMemcpy(&h_result, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    
    printf("Fibonacci(%u) = %u (Recursive method)\n", n, h_result);
    printf("Time taken: %f seconds\n", cpu_time_used);
    
    // Clean up
    hipFree(d_result);
    
    return 0;
}
