
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel for recursive Fibonacci
__device__ unsigned int fib_recursive_device(unsigned int n) {
    if (n <= 1) return n;
    return fib_recursive_device(n-1) + fib_recursive_device(n-2);
}

__global__ void fib_recursive_kernel(unsigned int n, unsigned int *result) {
    *result = fib_recursive_device(n);
}

// CUDA kernel for iterative Fibonacci
__global__ void fib_iterative_kernel(unsigned int n, unsigned int *result) {
    if (n <= 1) {
        *result = n;
        return;
    }
    
    unsigned int a = 0, b = 1, c;
    for (unsigned int i = 2; i <= n; i++) {
        c = a + b;
        a = b;
        b = c;
    }
    
    *result = b;
}

int main(int argc, char *argv[]) {
    // Default to calculating Fibonacci of 20 if no argument is provided
    unsigned int n = 20;
    if (argc > 1) {
        n = atoi(argv[1]);
    }
    
    // Allocate memory on the host and device
    unsigned int *d_result;
    unsigned int h_result;
    
    hipMalloc((void**)&d_result, sizeof(unsigned int));
    
    // Measure time for both methods
    clock_t start, end;
    double cpu_time_used;
    
    // Test iterative method first (faster)
    start = clock();
    fib_iterative_kernel<<<1, 1>>>(n, d_result);
    hipDeviceSynchronize();
    end = clock();
    
    hipMemcpy(&h_result, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    
    printf("Fibonacci(%u) = %u (Iterative method)\n", n, h_result);
    printf("Time taken: %f seconds\n", cpu_time_used);
    
    // For larger n values, the recursive method will be extremely slow or may crash
    if (n <= 25) {  // Set a reasonable limit for the recursive calculation
        start = clock();
        fib_recursive_kernel<<<1, 1>>>(n, d_result);
        hipDeviceSynchronize();
        end = clock();
        
        hipMemcpy(&h_result, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
        cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
        
        printf("Fibonacci(%u) = %u (Recursive method)\n", n, h_result);
        printf("Time taken: %f seconds\n", cpu_time_used);
    } else {
        printf("Skipping recursive calculation for n > 25 due to exponential time complexity\n");
    }
    
    // Free device memory
    hipFree(d_result);
    
    return 0;
}
